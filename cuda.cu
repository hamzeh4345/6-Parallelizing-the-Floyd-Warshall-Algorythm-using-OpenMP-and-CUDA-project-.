#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define INF 100

__global__ void floydWarshall(int *graph, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    for (int k = 0; k < n; k++)
    {
        int ik = i * n + k;
        int kj = k * n + j;
        int ij = i * n + j;
        if (graph[ij] > graph[ik] + graph[kj])
            graph[ij] = graph[ik] + graph[kj];
    }
}

int main(void)
{
    int n, i, j;
    printf("Enter the number of vertices: ");
    scanf("%d", &n);
    int *graph = (int *)malloc((long unsigned) n * n * sizeof(int));
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            if (i == j)
                graph[i * n + j] = 0;
            else
                graph[i * n + j] = INF;
        }
    }
    printf("Enter the edges: \n");
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("[%d][%d]: ", i, j);
            scanf("%d", &graph[i * n + j]);
        }
    }
    printf("The original graph is:\n");
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%d ", graph[i * n + j]);
        }
        printf("\n");
    }

    int *d_graph;
    hipMalloc((void **)&d_graph, n * n * sizeof(int));
    hipMemcpy(d_graph, graph, n * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    floydWarshall<<<gridSize, blockSize>>>(d_graph, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(graph, d_graph, n * n * sizeof(int), hipMemcpyDeviceToHost);

    printf("The shortest path matrix is:\n");
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%d ", graph[i * n + j]);
        }
        printf("\n");
    }

    printf("Execution time: %f ms\n", milliseconds);

    hipFree(d_graph);
    free(graph);

    return 0;
}
